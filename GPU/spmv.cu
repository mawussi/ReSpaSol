#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <mkl.h>
#include <mkl_spblas.h>
#include "loadMatrixMarket.h"


//#define FLOAT


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }

void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}


#define cusparseErrCheck(stat) { cusparseErrCheck_((stat), __FILE__, __LINE__); }
void cusparseErrCheck_(hipsparseStatus_t stat, const char *file, int line) {
  if (stat != HIPSPARSE_STATUS_SUCCESS) {
    fprintf(stderr, "CUSPARSE Error: %d %s %d\n", stat, file, line);
  }
}


int main(int argc, char **argv)
{

  if (argc < 2) {
    fprintf(
            stderr,
            "-- Usage examples --\n"
            "  %s inline_1.mtx type: run with inline_1 matrix in matrix market format\n",
            argv[0]);
    return -1;
  }


    CSR matrixA;
    int outputbase = 0;
    loadMatrixMarket(argv[1], &matrixA, outputbase, 0 /*transpose =false*/);
    
    
    const int A_num_rows = matrixA.n;
    const int A_num_cols = matrixA.n;
    const int A_num_nnz  = matrixA.nnz;
    
    int  *hA_csrOffsets = matrixA.rowptr;
    int  *hA_columns  = matrixA.colidx;  
    
    // index pointer on device
    int   *dA_csrOffsets, *dA_columns;
    
#ifdef FLOAT
    float alpha = (float)1.0;
    float beta = (float)0.0;
    float *hA_values = (float*)malloc(A_num_nnz*sizeof(float));
    #pragma omp parallel for
    for(int i =0; i < A_num_nnz; i++){
      hA_values[i] = (float) matrixA.values[i];   
    }
    float *Y        = (float*)malloc(A_num_rows*sizeof(float)); // result
    float *X        = (float*)malloc(A_num_cols*sizeof(float));
    #pragma omp parallel for
    for (int i = 0; i < A_num_cols; i++){ X[i] = (float) 1.0;}
    // device 
    float *dA_values;
    float *dX;
    float *dY;
#else
    double alpha = (double)1.0;
    double beta =  (double)0.0;
    double *hA_values = matrixA.values;
    double *Y        = (double*)malloc(A_num_rows*sizeof(double));
    double *X        = (double*)malloc(A_num_cols*sizeof(double));
     #pragma omp parallel for
    for (int i = 0; i < A_num_cols; i++){ X[i] = (double)1.0;}
    //device 
    double *dA_values;
    double *dX;
    double *dY;
#endif
    
    // Allocate device memory to store the sparse CSR representation of A
    cudaErrCheck(hipMalloc((void**) &dA_csrOffsets, (A_num_rows + 1) * sizeof(int)));
    cudaErrCheck(hipMalloc((void **)&dA_columns, A_num_nnz*sizeof(int)));
    
#ifdef FLOAT
    cudaErrCheck(hipMalloc((void **)&dA_values, A_num_nnz*sizeof(float)));
#else
    cudaErrCheck(hipMalloc((void **)&dA_values, A_num_nnz*sizeof(double)));
#endif 

    // Allocate device memory to store the X and Y
#ifdef FLOAT
    cudaErrCheck(hipMalloc((void **)&dX, A_num_cols*sizeof(float)));
    cudaErrCheck(hipMalloc((void **)&dY, A_num_rows*sizeof(float)));
#else
    cudaErrCheck(hipMalloc((void **)&dX, A_num_cols*sizeof(double)));
    cudaErrCheck(hipMalloc((void **)&dY, A_num_rows*sizeof(double)));
#endif 

    // transfer data to device 
    // Transfer the input vectors and dense matrix A to the device
    cudaErrCheck(hipMemcpy(dA_csrOffsets, hA_csrOffsets, (A_num_rows+1)*sizeof(int), hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(dA_columns, hA_columns, A_num_nnz*sizeof(int), hipMemcpyHostToDevice));
#ifdef FLOAT
    cudaErrCheck(hipMemcpy(dA_values, hA_values, A_num_nnz*sizeof(float),  hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(dX,      X         , A_num_cols*sizeof(float), hipMemcpyHostToDevice));
#else
    cudaErrCheck(hipMemcpy(dA_values, hA_values, A_num_nnz*sizeof(double),  hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(dX,      X,           A_num_cols*sizeof(double), hipMemcpyHostToDevice));
#endif


    // CUSPARSE APIs
    hipsparseHandle_t     handle = 0;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*  dBuffer    = NULL;
    size_t bufferSize = 0;
    cusparseErrCheck(hipsparseCreate(&handle));

    // Create sparse matrix A in CSR format
#ifdef FLOAT
    cusparseErrCheck( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_num_nnz,
					dA_csrOffsets, dA_columns, dA_values,
					HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // Create dense vector X
    cusparseErrCheck( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F) );
    
    // Create dense vector y
    cusparseErrCheck( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F) );

    // allocate an external buffer if needed
    cusparseErrCheck( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					      &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
					      HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) );

#else
    cusparseErrCheck( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_num_nnz,
					dA_csrOffsets, dA_columns, dA_values,
					HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    // Create dense vector X
    cusparseErrCheck( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_64F) );
    
    // Create dense vector y
    cusparseErrCheck( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_64F) );
    
    // allocate an external buffer if needed
    cusparseErrCheck( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					      &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
					      HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) );
#endif

    cudaErrCheck( hipMalloc(&dBuffer, bufferSize));
    
    
    // execute SpMV
    // Timing the solve 
    hipEvent_t start;
    hipEvent_t stop;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));
    
    int REPEAT = 50;
    float times [REPEAT];
    for (int i = 0; i < REPEAT; i++) {
      hipEventRecord(start);
#ifdef FLOAT
      cusparseErrCheck( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				     &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
				     HIPSPARSE_MV_ALG_DEFAULT, dBuffer));
      
#else
      cusparseErrCheck( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				     &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
				     HIPSPARSE_MV_ALG_DEFAULT, dBuffer) );
#endif	 
      
      hipEventRecord(stop);  
      hipEventSynchronize(stop);
      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start, stop);
      
      times[i] = milliseconds;
    }
    
    float sum_times = 0.0f;
    for (int i = 0; i < REPEAT; i++) {
      sum_times += times[i];
    }
    
#ifdef FLOAT
    printf ("SINGLE PRECISION SPMV ");
#else
    printf ("DOUBLE PRECISION SPMV ");
#endif
    printf ("solve time (microseconds) = %f\n", (sum_times/REPEAT)*1000);


    // device result check
#ifdef FLOAT
    cudaErrCheck( hipMemcpy(Y, dY, A_num_rows * sizeof(float),
    hipMemcpyDeviceToHost));
    
#else
    cudaErrCheck( hipMemcpy(Y, dY, A_num_rows * sizeof(double),
    hipMemcpyDeviceToHost) )
#endif


    // use mkl to check result
    sparse_matrix_t mklA;
    sparse_status_t stat;

#ifdef FLOAT
    stat = mkl_sparse_s_create_csr(&mklA, SPARSE_INDEX_BASE_ZERO, A_num_rows, A_num_cols,
				   hA_csrOffsets, hA_csrOffsets + 1,
				   hA_columns, hA_values);
#else
    stat = mkl_sparse_d_create_csr(&mklA, SPARSE_INDEX_BASE_ZERO, A_num_rows, A_num_cols,
				   hA_csrOffsets, hA_csrOffsets + 1,
				   hA_columns, hA_values);
#endif
    if (SPARSE_STATUS_SUCCESS != stat) {
      fprintf(stderr, "Failed to create mkl csr\n");
      return -1;
    }
#ifdef FLOAT    
    float *result = (float*)malloc(sizeof(float)*A_num_rows);
    float error = 0;
#else
    double *result = (double*)malloc(sizeof(double)*A_num_rows);
    double error = 0;
#endif
    
    matrix_descr descA;
    descA.type = SPARSE_MATRIX_TYPE_GENERAL;
    descA.diag = SPARSE_DIAG_NON_UNIT;

#ifdef FLOAT
    mkl_sparse_s_mv(SPARSE_OPERATION_NON_TRANSPOSE, 1, mklA, descA, X, 0, result);
#else
    mkl_sparse_d_mv(SPARSE_OPERATION_NON_TRANSPOSE, 1, mklA, descA, X, 0, result);
#endif

  #pragma omp parallel for
  for(int i = 0; i < A_num_rows; i++) {
    error += abs(result[i] - Y[i]);
  }
  printf ("Error= %e\n", error/A_num_cols);

    // destroy matrix/vector descriptors    
    cudaErrCheck(hipEventDestroy(start));             
    cudaErrCheck(hipEventDestroy(stop));
    free(hA_values);
    free(hA_csrOffsets);
    free(hA_columns);
    free(X);
    free(Y);
    free(result);
    cudaErrCheck(hipFree(dY));
    cudaErrCheck(hipFree(dX));
    cudaErrCheck(hipFree(dA_values));
    cudaErrCheck(hipFree(dA_csrOffsets));
    cudaErrCheck(hipFree(dA_columns));
    cudaErrCheck(hipFree(dBuffer));
    

    cusparseErrCheck( hipsparseDestroySpMat(matA));
    cusparseErrCheck( hipsparseDestroyDnVec(vecX));
    cusparseErrCheck( hipsparseDestroyDnVec(vecY));
    cusparseErrCheck( hipsparseDestroy(handle) );
    return 0;
}
