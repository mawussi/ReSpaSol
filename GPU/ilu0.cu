#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "loadMatrixMarket.h"

#define FLOAT


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }

void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}


#define cusparseErrCheck(stat) { cusparseErrCheck_((stat), __FILE__, __LINE__); }
void cusparseErrCheck_(hipsparseStatus_t stat, const char *file, int line) {
  if (stat != HIPSPARSE_STATUS_SUCCESS) {
    fprintf(stderr, "CUSPARSE Error: %d %s %d\n", stat, file, line);
  }
}


int main(int argc, char **argv)
{

  if (argc < 2) {
    fprintf(
            stderr,
            "-- Usage examples --\n"
            "  %s inline_1.mtx type: run with inline_1 matrix in matrix market format\n",
            argv[0]);
    return -1;
  }


    CSR matrixA;
    int outputbase = 0;
    loadMatrixMarket(argv[1], &matrixA, outputbase, 0 /*transpose =false*/);
    
    
    int             n  = matrixA.n;
    int *CsrRowPtrA =  matrixA.rowptr;
    int *CsrColIndA =  matrixA.colidx; 
    
    // index pointer on device
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    
#ifdef FLOAT
    float alpha = (float)1.0;
    float *CsrValA = (float*)malloc(matrixA.nnz*sizeof(float));
    for(int i =0; i < matrixA.nnz; i++){
      CsrValA[i] = (float) matrixA.values[i];   
    }
    float *Y        = (float*)malloc(n*sizeof(float));
    float *X        = (float*)malloc(n*sizeof(float));
    for (int i = 0; i < n; i++) X[i] = (float) 1.0;
    // device 
    float *dCsrValA;
    float *dX;
    float *dZ; // intermediate solution Lz =b
    float *dY;
#else
    double alpha = (double)1.0;
    double *CsrValA = matrixA.values;
    double *Y        = (double*)malloc(n*sizeof(double));
    double *X        = (double*)malloc(n*sizeof(double));
    for (int i = 0; i < n; i++) X[i] = (double)1.0;
    //device 
    double *dCsrValA;
    double *dX;
    double *dZ; // intermediate solution Lz =b
    double *dY;
#endif
    
    hipsparseHandle_t handle = 0;
    
    // Create the cuSPARSE handle
     cusparseErrCheck(hipsparseCreate(&handle));


    // Allocate device memory to store the sparse CSR representation of A
    cudaErrCheck(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (n+1)));
    cudaErrCheck(hipMalloc((void **)&dCsrColIndA, sizeof(int) * matrixA.nnz));
    
#ifdef FLOAT
    cudaErrCheck(hipMalloc((void **)&dCsrValA, sizeof(float) * matrixA.nnz));
#else
    cudaErrCheck(hipMalloc((void **)&dCsrValA, sizeof(double) * matrixA.nnz));
#endif 

    // Allocate device memory to store the X and Y
#ifdef FLOAT
    cudaErrCheck(hipMalloc((void **)&dX, sizeof(float) * n));
    cudaErrCheck(hipMalloc((void **)&dY,    sizeof(float) * n));
    cudaErrCheck(hipMalloc((void **)&dZ,    sizeof(float) * n));
#else
    cudaErrCheck(hipMalloc((void **)&dX, sizeof(double) * n));
    cudaErrCheck(hipMalloc((void **)&dY,    sizeof(double) * n));
    cudaErrCheck(hipMalloc((void **)&dZ,    sizeof(double) * n));
#endif 

    // transfer data to device 
    // Transfer the input vectors and dense matrix A to the device
    cudaErrCheck(hipMemcpy(dCsrRowPtrA, CsrRowPtrA, sizeof(int) * (n+1), hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(dCsrColIndA, CsrColIndA, sizeof(int) * matrixA.nnz, hipMemcpyHostToDevice));
#ifdef FLOAT
    cudaErrCheck(hipMemcpy(dCsrValA,  CsrValA, sizeof(float) * matrixA.nnz, hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(dX,      X,     sizeof(float) * n,           hipMemcpyHostToDevice));
#else
    cudaErrCheck(hipMemcpy(dCsrValA, CsrValA, sizeof(double) * matrixA.nnz, hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(dX,     X,     sizeof(double) * n,          hipMemcpyHostToDevice));
#endif


    // Create descriptor A
    hipsparseMatDescr_t desc_A = 0;
    cusparseErrCheck(hipsparseCreateMatDescr(&desc_A));
    cusparseErrCheck(hipsparseSetMatType(desc_A, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseErrCheck(hipsparseSetMatIndexBase(desc_A, HIPSPARSE_INDEX_BASE_ZERO));

    // create descriptor L
    hipsparseMatDescr_t desc_L = 0;
    hipsparseCreateMatDescr(&desc_L);
    hipsparseSetMatIndexBase(desc_L, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(desc_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseMatDescr_t desc_U = 0;
    hipsparseCreateMatDescr(&desc_U);
    hipsparseSetMatIndexBase(desc_U, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(desc_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(desc_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    //  Create a empty info structure
    csrilu02Info_t info_A  = 0;
    csrsv2Info_t   info_L  = 0;
    csrsv2Info_t   info_U = 0;

    hipsparseCreateCsrilu02Info(&info_A);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    //  Query how much memory used in csric02 and csrsv2, and allocate the buffer
    int pBufferSize_A;
    int pBufferSize_L;
    int pBufferSize_U;


    // Timing variables
    hipEvent_t start;
    hipEvent_t stop;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));


#ifdef FLOAT
    hipsparseScsrilu02_bufferSize(handle, n, matrixA.nnz,
				desc_A, dCsrValA, dCsrRowPtrA, dCsrColIndA,
				info_A, &pBufferSize_A);

    hipsparseScsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz,
			       desc_L, dCsrValA, dCsrRowPtrA, dCsrColIndA,
			       info_L, &pBufferSize_L);

    hipsparseScsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz,
			       desc_U, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_U,&pBufferSize_U);
#else 
    hipsparseDcsrilu02_bufferSize(handle, n, matrixA.nnz,
				desc_A, dCsrValA, dCsrRowPtrA, dCsrColIndA,
				info_A, &pBufferSize_A);

    hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz,
			       desc_L, dCsrValA, dCsrRowPtrA, dCsrColIndA,
			       info_L, &pBufferSize_L);

    hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz,
			       desc_U, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_U,&pBufferSize_U);
#endif


    int   pBufferSize = max(pBufferSize_A, max(pBufferSize_L, pBufferSize_U));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    void *pBuffer = 0;
    hipMalloc((void**)&pBuffer, pBufferSize);

    // Timing the analysis 
    hipEventRecord(start);

    // Perform analysis of ILU0 on A
    const hipsparseSolvePolicy_t policy_A  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;

#ifdef FLOAT
    hipsparseScsrilu02_analysis(handle, n, matrixA.nnz, desc_A,
			       dCsrValA, dCsrRowPtrA, dCsrColIndA, info_A,
			       policy_A, pBuffer);
#else
    hipsparseDcsrilu02_analysis(handle, n, matrixA.nnz, desc_A,
			      dCsrValA, dCsrRowPtrA, dCsrColIndA, info_A,
			      policy_A, pBuffer);
#endif 

    
    hipEventRecord(stop);  
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float time_symbolique  = milliseconds;



    int structural_zero;
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
      printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
      return 0;
    }

    // Perform analysis of triangular solve on L
    const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;

#ifdef FLOAT
    hipsparseScsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz, desc_L,
			     dCsrValA, dCsrRowPtrA, dCsrColIndA,
			     info_L, policy_L, pBuffer);
#else
    hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz, desc_L,
			     dCsrValA, dCsrRowPtrA, dCsrColIndA,
			     info_L, policy_L, pBuffer);
#endif 

    // Perform analysis of triangular solve on U
    const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

#ifdef FLOAT
    hipsparseScsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz, desc_L,
			     dCsrValA, dCsrRowPtrA, dCsrColIndA,
			     info_U, policy_U, pBuffer);
#else
    hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz, desc_L,
			     dCsrValA, dCsrRowPtrA, dCsrColIndA,
			     info_U, policy_U, pBuffer);
#endif 
    
    


    // Numerical factorization
    int numerical_zero;

    // Timing the numerical factorization 
    hipEventRecord(start);

#ifdef FLOAT
    hipsparseScsrilu02(handle, n, matrixA.nnz, desc_A,
		      dCsrValA, dCsrRowPtrA, dCsrColIndA, info_A, policy_A, pBuffer);
#else
    hipsparseDcsrilu02(handle, n, matrixA.nnz, desc_A,
		     dCsrValA, dCsrRowPtrA, dCsrColIndA, info_A, policy_A, pBuffer);
#endif 

    hipEventRecord(stop);  
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float time_numeric  = milliseconds;


    status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
      printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
      return 0;
    }

    hipEventRecord(start);
#ifdef FLOAT
    //  Solve L*z = x
    hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz, &alpha, desc_L,
			  dCsrValA, dCsrRowPtrA, dCsrColIndA, info_L,
			  dX, dZ, policy_L, pBuffer);
    // Solve L'*y = z
    hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_TRANSPOSE, n, matrixA.nnz, &alpha, desc_L,
			  dCsrValA, dCsrRowPtrA, dCsrColIndA, info_U,
			  dZ, dY, policy_U, pBuffer);
#else
    //  Solve L*z = x
    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, matrixA.nnz, &alpha, desc_L,
			  dCsrValA, dCsrRowPtrA, dCsrColIndA, info_L,
			  dX, dZ, policy_L, pBuffer);
    // Solve L'*y = z
    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_TRANSPOSE, n, matrixA.nnz, &alpha, desc_L,
			  dCsrValA, dCsrRowPtrA, dCsrColIndA, info_U,
			  dZ, dY, policy_U, pBuffer);
#endif
    
    hipEventRecord(stop);  
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    float time_solve  = milliseconds;
    
#ifdef FLOAT
    printf ("SINGLE PRECISION SOLVE IN  MILLISECONDS\n ");
#else
    printf ("DOUBLE PRECISION SOLVE IN  MILLISECONDS\n ");
#endif
    printf ("Symbolic = %f\n Numeric = %f \n Symbolic+ Numeric = %f\n Solve = %f\n", time_symbolique, time_numeric, time_symbolique + time_numeric,  time_solve);
 
    cudaErrCheck(hipEventDestroy(start));             
    cudaErrCheck(hipEventDestroy(stop));
    free(CsrValA);
    free(CsrRowPtrA);
    free(CsrColIndA);
    free(X);
    free(Y);
    cudaErrCheck(hipFree(dY));
    cudaErrCheck(hipFree(dX));
    cudaErrCheck(hipFree(dCsrValA));
    cudaErrCheck(hipFree(dCsrRowPtrA));
    cudaErrCheck(hipFree(dCsrColIndA));
    cudaErrCheck(hipFree(pBuffer));
    
    hipsparseDestroyCsrilu02Info(info_A);
    hipsparseDestroyCsrsv2Info(info_L);
    hipsparseDestroyCsrsv2Info(info_U);

    cusparseErrCheck(hipsparseDestroyMatDescr(desc_A));
    cusparseErrCheck(hipsparseDestroyMatDescr(desc_L));
    cusparseErrCheck(hipsparseDestroyMatDescr(desc_U));
    cusparseErrCheck(hipsparseDestroy(handle));
    return 0;
}
